#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
#define TILE_WIDTH 16 // number of threads per block
 
// kernel function for convolution
__global__ void convolve(float* N, float* M, float* P, int width, int mask_width) {
    // calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
 
    // initialize output element to zero
    float P_val = 0;
 
    // loop over the mask array
    for (int j = 0; j < mask_width; j++) {
        // calculate the index of the input element
        int k = i - (mask_width / 2) + j;
 
        // check if the index is within bounds
        if (k >= 0 && k < width) {
            // accumulate the product of input and mask elements
            P_val += N[k] * M[j];
        }
    }
 
    // store the output element in the output array
    P[i] = P_val;
}
 
int main() {
 
    // initialize input array N
    float N[] = { 1, 2, 3, 4, 5 };
 
    // initialize mask array M
    float M[] = { 0.2, 0.2, 0.2, 0.2, 0.2 };
 
    // get the sizes of the arrays
    int width = sizeof(N) / sizeof(float);
    int mask_width = sizeof(M) / sizeof(float);
 
    // allocate memory for output array P on host
    float* P = (float*)malloc(width * sizeof(float));
 
    // allocate memory for arrays on device
    float* d_N, * d_M, * d_P;
    hipMalloc((void**)&d_N, width * sizeof(float));
    hipMalloc((void**)&d_M, mask_width * sizeof(float));
    hipMalloc((void**)&d_P, width * sizeof(float));
 
    // copy arrays from host to device
    hipMemcpy(d_N, N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(float), hipMemcpyHostToDevice);
 
    // calculate number of blocks needed
    int num_blocks = ceil((float)width / TILE_WIDTH);
 
    // launch kernel function
    convolve << <num_blocks, TILE_WIDTH >> > (d_N, d_M, d_P, width, mask_width);
 
    // copy output array from device to host
    hipMemcpy(P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);
 
    printf("Input array P:\n");
    for (int i = 0; i < width; i++) {
        printf("%f ", N[i]);
    }
    printf("\n");
    printf("Mask array P:\n");
    for (int i = 0; i < width; i++) {
        printf("%f ", P[i]);
    }
    printf("\n");
 
    // print output array
    printf("Output array P:\n");
    for (int i = 0; i < width; i++) {
        printf("%f ", P[i]);
    }
    printf("\n");
 
    // free memory on host and device
    free(P);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);
 
    return 0;
}
