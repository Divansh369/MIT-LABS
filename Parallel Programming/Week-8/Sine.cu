
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void computeSine(float *input, float *output, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        output[tid] = sinf(input[tid]);
    }
}

int main() {
    int size = 10; // Size of the input array
    size_t bytes = size * sizeof(float);

    // Allocate memory for the host arrays
    float *h_input = (float*)malloc(bytes);
    float *h_output = (float*)malloc(bytes);

    // Initialize the input array with angles in radians
    for (int i = 0; i < size; ++i) {
        h_input[i] = 200*i; // Increment angle by 0.01 radians
    }

    // Allocate memory for the device arrays
    float *d_input, *d_output;
    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    // Copy the input array from host to device
    hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    computeSine<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    // Copy the result array from device to host
    hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < size; ++i) {
        printf("sin(%f radians) = %f\n", h_input[i], h_output[i]);
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}
