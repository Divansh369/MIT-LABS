
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_SENTENCE_LENGTH 1000
#define MAX_WORD_LENGTH 100

__global__ void countWordOccurrences(char *sentence, char *word, int *count, int sentenceLength, int wordLength) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // int wordCount = 0;
    if (tid < sentenceLength - wordLength + 1) {
        int match = 1;
        for (int i = 0; i < wordLength; ++i) {
            if (sentence[tid + i] != word[i]) {
                match = 0;
                break;
            }
        }
        if (match) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char sentence[MAX_SENTENCE_LENGTH];
    char word[MAX_WORD_LENGTH];
    int sentenceLength, wordLength;

    printf("Enter a sentence: ");
    fgets(sentence, MAX_SENTENCE_LENGTH, stdin);
    sentenceLength = strlen(sentence);
    if (sentence[sentenceLength - 1] == '\n') {
        sentence[sentenceLength - 1] = '\0';
        sentenceLength--;
    }

    printf("Enter the word to count: ");
    scanf("%s", word);
    wordLength = strlen(word);

    char *d_sentence, *d_word;
    int *d_count;
    int count = 0;
    hipMalloc(&d_sentence, sentenceLength * sizeof(char));
    hipMalloc(&d_word, wordLength * sizeof(char));
    hipMalloc(&d_count, sizeof(int));

    hipMemcpy(d_sentence, sentence, sentenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (sentenceLength + threadsPerBlock - 1) / threadsPerBlock;

    countWordOccurrences<<<blocksPerGrid, threadsPerBlock>>>(d_sentence, d_word, d_count, sentenceLength, wordLength);

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("Number of occurrences of '%s' in the sentence: %d\n", word, count);

    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
