
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_STRING_LENGTH 1000

__global__ void repeatString(char *S, char *RS, int length, int S_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        RS[tid] = S[tid % S_length];
    }
}

int main() {
    char S[MAX_STRING_LENGTH];
    char RS[MAX_STRING_LENGTH * 3]; // Assuming the output string can be at most 3 times the length of the input string
    int length;

    printf("Enter a string: ");
    fgets(S, MAX_STRING_LENGTH, stdin);
    length = strlen(S);
    if (S[length - 1] == '\n') {
        S[length - 1] = '\0';
        length--;
    }

    int S_length = length; // Store the length of input string
    length *= 3; // Adjust length for output string

    char *d_S, *d_RS;
    hipMalloc(&d_S, S_length * sizeof(char));
    hipMalloc(&d_RS, length * sizeof(char));

    hipMemcpy(d_S, S, S_length * sizeof(char), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (length + threadsPerBlock - 1) / threadsPerBlock;

    repeatString<<<blocksPerGrid, threadsPerBlock>>>(d_S, d_RS, length, S_length);

    hipMemcpy(RS, d_RS, length * sizeof(char), hipMemcpyDeviceToHost);

    printf("Input String S: %s\n", S);
    printf("Output String RS: %s\n", RS);

    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
