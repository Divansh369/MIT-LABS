#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
 
// CUDA kernel to add two vectors
__global__ void vecAddKernel(int* A, int* B, int* C, int n) {
    // Get the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the thread is within the vector range
    if (id < n) {
        // Add the corresponding elements of A and B
        C[id] = A[id] + B[id];
    }
}
 
 
// Function to add two vectors using CUDA
void vecAdd(int* A, int* B, int* C, int n) {
    // Allocate device memory for the vectors
    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, n * sizeof(int));
    hipMalloc(&d_B, n * sizeof(int));
    hipMalloc(&d_C, n * sizeof(int));
 
 
    // Copy the vectors from host to device
    hipMemcpy(d_A, A, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * sizeof(int), hipMemcpyHostToDevice);
 
 
    // Define the block size and grid size
    int blockSize = 256; // Number of threads per block
    int gridSize = (n + 256 - 1) / n; // Number of blocks per grid
 
 
    // Launch the kernel with the specified configuration
    vecAddKernel << <gridSize, blockSize >> > (d_A, d_B, d_C, n);
 
 
    // Copy the result vector from device to host
    hipMemcpy(C, d_C, n * sizeof(int), hipMemcpyDeviceToHost);
 
 
    // Free the device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
 
 
// Main function to test the vector addition
int main() {
    // Get the vector length from the user
    int n;
    printf("Enter the vector length: ");
    scanf("%d", &n);
 
 
    // Allocate host memory for the vectors
    int* A = (int*)malloc(n * sizeof(int));
    int* B = (int*)malloc(n * sizeof(int));
    int* C = (int*)malloc(n * sizeof(int));
 
 
    // Initialize the vectors with random values
    for (int i = 0; i < n; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
        C[i] = 0;
    }
 
 
    // Add the vectors using CUDA
    vecAdd(A, B, C, n);
    printf("\nThe first vector is:\n");
    for (int i = 0; i < n; i++) {
        printf("%d\t", A[i]);
    }
    printf("\nThe second vector is:\n");
    for (int i = 0; i < n; i++) {
        printf("%d\t", B[i]);
    }
    // Print the result vector
    printf("\n\nThe result vector is:\n");
    for (int i = 0; i < n; i++) {
        printf("%d\t", C[i]);
    }
 
    printf("\n");
    // Free the host memory
    free(A);
    free(B);
    free(C);
 
 
    return 0;
}
