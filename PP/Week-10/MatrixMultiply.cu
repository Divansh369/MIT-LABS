
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MAX_SIZE 10 // Maximum size of the matrices

// Kernel to multiply two matrices where each row of the resultant matrix is computed by one thread
__global__ void multiplyMatrixRows(int *A, int *B, int *C, int size) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < size && col < size) {
        int sum = 0;
        for (int k = 0; k < size; ++k) {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;
    }
}

// Kernel to multiply two matrices where each column of the resultant matrix is computed by one thread
__global__ void multiplyMatrixColumns(int *A, int *B, int *C, int size) {
    int row = threadIdx.x;
    int col = blockIdx.x;
    if (row < size && col < size) {
        int sum = 0;
        for (int k = 0; k < size; ++k) {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;
    }
}

// Kernel to multiply two matrices where each element of the resultant matrix is computed by one thread
__global__ void multiplyMatrixElements(int *A, int *B, int *C, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int row = index / size;
    int col = index % size;
    if (row < size && col < size) {
        int sum = 0;
        for (int k = 0; k < size; ++k) {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[index] = sum;
    }
}

// Function to print a matrix
void printMatrix(int *matrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            printf("%d ", matrix[i * size + j]);
        }
        printf("\n");
    }
}

int main() {
    int size;
    printf("Enter the size of the matrices (maximum %d): ", MAX_SIZE);
    scanf("%d", &size);
    if (size <= 0 || size > MAX_SIZE) {
        printf("Invalid size!\n");
        return 1;
    }

    int *A, *B, *C;
    size_t matrixSize = size * size * sizeof(int);

    // Allocate memory for matrices on the host
    A = (int *)malloc(matrixSize);
    B = (int *)malloc(matrixSize);
    C = (int *)malloc(matrixSize);

    // Initialize random number generator
    srand(time(NULL));

    // Generate random matrices A and B
    for (int i = 0; i < size * size; ++i) {
        A[i] = rand() % 10; // Generate a random number between 0 and 9
        B[i] = rand() % 10;
    }

    // Print matrices A and B
    printf("Matrix A:\n");
    printMatrix(A, size);
    printf("\nMatrix B:\n");
    printMatrix(B, size);

    int *d_A, *d_B, *d_C;
    // Allocate memory for matrices on the device
    hipMalloc(&d_A, matrixSize);
    hipMalloc(&d_B, matrixSize);
    hipMalloc(&d_C, matrixSize);

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, matrixSize, hipMemcpyHostToDevice);

    // Launch kernels to multiply matrices using different thread specifications
    // Uncomment the desired kernel call

    // multiplyMatrixRows<<<size, size>>>(d_A, d_B, d_C, size); // Each row of resultant matrix computed by one thread
    // multiplyMatrixColumns<<<size, size>>>(d_A, d_B, d_C, size); // Each column of resultant matrix computed by one thread
    multiplyMatrixElements<<<(size * size + 255) / 256, 256>>>(d_A, d_B, d_C, size); // Each element of resultant matrix computed by one thread

    // Copy resultant matrix C from device to host
    hipMemcpy(C, d_C, matrixSize, hipMemcpyDeviceToHost);

    // Print resultant matrix C
    printf("\nResultant Matrix:\n");
    printMatrix(C, size);

    // Free memory allocated on the device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free memory allocated on the host
    free(A);
    free(B);
    free(C);

    return 0;
}
